#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_cooperative_groups.h>

#include <iostream>
#include <vector>
#include <cassert>

using namespace cooperative_groups;
using namespace std;

#define CUDA_CHECK(expr) do {\
    hipError_t err = (expr);\
    if (err != hipSuccess) {\
        std::cerr << "CUDA ERROR: " << __FILE__ << ":" << __LINE__ << ": " << #expr << " <" << hipGetErrorName(err) << "> " << hipGetErrorString(err) << "\n"; \
        abort(); \
    }\
} while(0)


extern __shared__ void* sharedBuffer[];
#define SHARED_BUFFER_OWNER (((int*)sharedBuffer)[0])
#define SHARED_BUFFER_DATA (void*)(&((int*)sharedBuffer)[1])


__device__ void p2pSendBlock(void* data, size_t dataSize, int srcThread, int dstThread) {
    int rank = this_thread_block().thread_rank();
    
    if (srcThread != rank && dstThread != rank) return;
    
    bool done = false;
    while (!done) {
        if (srcThread == rank) {
            if (atomicCAS(&SHARED_BUFFER_OWNER, -1, srcThread) == srcThread) {
                memcpy(SHARED_BUFFER_DATA, data, dataSize);
                // next line prevents other threads from using data before
                // current thread entirely copied it
                __threadfence_block();
                SHARED_BUFFER_OWNER = dstThread;
                // current thread allowed to go out of loop
                // only if other threads notified that he changed ownership
                __threadfence_block();
                done = true;
            }
        } else {
            if (SHARED_BUFFER_OWNER == dstThread) {
                memcpy(data, SHARED_BUFFER_DATA, dataSize);
                __threadfence_block();
                SHARED_BUFFER_OWNER = -1;
                __threadfence_block();
                done = true;
            }
        }
    }
}

__global__ void kernelBenchmarkBlock(size_t dataSize, char* deviceSrcData, char* deviceDstData, int peakClkKHz) {
    
    coalesced_group warp = coalesced_threads();
    assert(warp.size() == 32);
    if (warp.thread_rank() != 0) return;
    
    int rank = this_thread_block().thread_rank();
    
    if (rank == 0) {
        SHARED_BUFFER_OWNER = 0;
    }
    this_thread_block().sync();
    void* data = nullptr;
    if (rank == 0) {
        data = deviceSrcData;
    } else {
        data = deviceDstData;
    }
    
    int repetitions = 100;
    
    auto t1 = clock64();

    
    
    for (int r = 0; r < repetitions; r++) {
        p2pSendBlock(data, dataSize, /*srcThread*/ 0, /*dstThread*/ 32);
        p2pSendBlock(data, dataSize, /*srcThread*/ 32, /*dstThread*/ 0);
    }
    
    auto t2 = clock64();
    
    if (this_thread_block().thread_rank() == 0) {
        double totalTime = (t2 - t1) * 0.001 / peakClkKHz;
        double timePerSend = totalTime / repetitions / 2;
        double bandwidth = dataSize / timePerSend;
        printf("dataSize = %d B, time = %lg us, bandwidth = %lg MB/s \n", int(dataSize), timePerSend * 1e6, bandwidth / 1e6);
    }
}

int main() {
    int deviceCount = -1;
    CUDA_CHECK(hipGetDeviceCount(&deviceCount));
    assert(deviceCount >= 1);
    
    int peakClkKHz = -1;
    CUDA_CHECK(hipDeviceGetAttribute(&peakClkKHz, hipDeviceAttributeClockRate, /*device = */0));
    assert(peakClkKHz > 0);
    
    int sharedMemPerBlock = -1;
    CUDA_CHECK(hipDeviceGetAttribute(&sharedMemPerBlock, hipDeviceAttributeMaxSharedMemoryPerBlock, /*device = */0));
    // there is also sharedMemPerMultiprocessor
    assert(sharedMemPerBlock > 0);
    
    
    
    for (size_t dataSize = 1; dataSize < sharedMemPerBlock - sizeof(int); dataSize *= 2) {
        
        char* deviceSrcData = nullptr;
        CUDA_CHECK(hipMalloc(&deviceSrcData, dataSize));
        
        char* deviceDstData = nullptr;
        CUDA_CHECK(hipMalloc(&deviceDstData, dataSize));
        
        char* hostSrcData = (char*) malloc(dataSize);
        assert(hostSrcData);
        char* hostDstData = (char*) malloc(dataSize);
        assert(hostDstData);
        
        for (int i = 0; i < dataSize; i++) hostSrcData[i] = 17;
        
        CUDA_CHECK(hipMemcpy(deviceSrcData, hostSrcData, dataSize, hipMemcpyHostToDevice));
        
        dim3 blocksPerGrid = 1;
        dim3 threadsPerBlock = 64;
        size_t sharedBufferSize = dataSize + sizeof(int);
        kernelBenchmarkBlock<<<blocksPerGrid, threadsPerBlock, sharedBufferSize>>>(dataSize, deviceSrcData, deviceDstData, peakClkKHz);
        
        CUDA_CHECK(hipPeekAtLastError());
        
        CUDA_CHECK(hipDeviceSynchronize());
        
        CUDA_CHECK(hipMemcpy(hostDstData, deviceDstData, dataSize, hipMemcpyDeviceToHost));
        
        for (int i = 0; i < dataSize; i++) {
            if (hostDstData[i] != 17) {
                printf("Incorrect data!");
                abort();
            }
        }
        
        free(hostDstData);
        free(hostSrcData);
        
        CUDA_CHECK(hipFree(deviceDstData));
        CUDA_CHECK(hipFree(deviceSrcData));
    }
    
    printf("Exit\n");
    
    return 0;
}
