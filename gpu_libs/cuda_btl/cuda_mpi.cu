#include "hip/hip_runtime.h"
#include "cuda_mpi.cuh"

#ifdef ENABLE_GPU_MPI_LOG
#define LOG(fmt, ...) printf("Thread %d " __FILE__ ":%d " fmt "\n", cg::this_grid().thread_rank(), __LINE__,## __VA_ARGS__)
#else
#define LOG(fmt, ...)
#endif

#define $ LOG("STILL ALIVE!");

__device__ void memcpy_volatile(volatile void *dst, volatile void *src, size_t n)
{
    volatile char *d = (volatile char*) dst;
    volatile char *s = (volatile char*) src;
    for (size_t i = 0; i < n; i++) {
        d[i] = s[i];
    }
}

template <typename T>
class ScopeGuard {
public:
    __host__ __device__ ScopeGuard(T func) : run(true), func(func) {}
    __host__ __device__ ScopeGuard(ScopeGuard<T>&& rhs)
        : run(rhs.run)
        , func(std::move(rhs.func))
    { rhs.run = false; }
    __host__ __device__ ~ScopeGuard() { if (run) func(); }
    __host__ __device__ void commit() { run = false; }
private:
    ScopeGuard(const ScopeGuard<T>& rhs) = delete;
    void operator=(const ScopeGuard<T>& rhs) = delete;
    
    bool run;
    T func;
};

template <typename T>
__host__ __device__ ScopeGuard<T> makeScopeGuard(T func) {
    return ScopeGuard<T>(func);
}

namespace CudaMPI {

// this pointer should be initialized before executing any other functions
// size of this array should be equal to the number of launched threads
// on this device
__device__ SharedState* gSharedState = nullptr;

__device__ SharedState& sharedState() {
    assert(gSharedState != nullptr);
    return *gSharedState;
};

__device__ void setSharedState(SharedState* sharedState) {
    if (cg::this_grid().thread_rank() == 0) {
        VOLATILE(gSharedState) = sharedState;
    }
    cg::this_grid().sync();
}

__device__ PendingOperation* ThreadPrivateState::allocatePendingOperation() {
    if (pendingOperations.full()) return nullptr;
    int insertedIndex = pendingOperations.push(PendingOperation());
    return &pendingOperations.get(insertedIndex);
}

__device__ ThreadPrivateState* gThreadLocalState = nullptr;

__device__ ThreadPrivateState& threadPrivateState() {
    assert(gThreadLocalState != nullptr);
    int gridIdx = cg::this_grid().thread_rank();
    return gThreadLocalState[gridIdx];
}

__device__ ThreadPrivateState::Holder::Holder(const Context& ctx) {
    LOG("initializeThreadPrivateState");
    if (0 == cg::this_grid().thread_rank()) {
        gThreadLocalState = (ThreadPrivateState*)malloc(cg::this_grid().size() * sizeof(ThreadPrivateState));
        assert(gThreadLocalState);
        __threadfence_system();
    }
    cg::this_grid().sync();
    assert(gThreadLocalState);
    new (&threadPrivateState()) ThreadPrivateState(ctx);
}

__device__ ThreadPrivateState::Holder::~Holder() {
    LOG("destroyThreadPrivateState");
    threadPrivateState().~ThreadPrivateState();
    cg::this_grid().sync();
    if (0 == cg::this_grid().thread_rank()) {
        free(gThreadLocalState);
    }
}

__device__ PendingOperation* isend(int dst, const void* data, int count, int comm, int tag) {
    LOG("isend");
    PendingOperation* po = threadPrivateState().allocatePendingOperation();
    while (!po) {
        po = threadPrivateState().allocatePendingOperation();
        printf("WARNING: Pending operations limit is reached in isend, this can cause a deadlock\n");
        progress();
    }

    po->type = PendingOperation::Type::SEND;
    po->state = PendingOperation::State::STARTED;
    po->fragment = nullptr;
    po->otherThread = dst;
    po->data = (void*) data;
    po->count = count;
    po->comm = comm;
    po->tag = tag;
    po->canBeFreed = false;
//     po->unused = false;

    progress();

    return po;
}

__device__ PendingOperation* irecv(int src, void* data, int count, int comm, int tag) {
    LOG("irecv");

    PendingOperation* po = threadPrivateState().allocatePendingOperation();
    while (!po) {
        po = threadPrivateState().allocatePendingOperation();
        LOG("WARNING: Pending operations limit is reached in irecv, this can cause a deadlock\n");
        progress();
    }

    po->type = PendingOperation::Type::RECV;
    po->state = PendingOperation::State::STARTED;
    po->fragment = nullptr;
    po->otherThread = src;
    po->data = data;
    po->count = count;
    po->comm = comm;
    po->tag = tag;
    po->canBeFreed = false;
//     po->unused = false;

    progress();

    return po;
}

__device__ void progressCompletedRecv(PendingOperation& recv) {
    LOG("progressCompletedRecv %p", &recv);

    if (recv.fragment) {
        LOG("unlocking memory fragment %p", recv.fragment);
        recv.fragment->memoryLock.unlock();
        recv.fragment = nullptr;
    }
    
    if (recv.canBeFreed) {
        LOG("freeing local recv operation");
        threadPrivateState().getPendingOperations().pop(&recv);
    }
}

__device__ void progressCompletedSend(PendingOperation& send) {
    LOG("progressCompletedSend %p", &send);

    if (send.canBeFreed) {
        LOG("freeing local send operation");
        threadPrivateState().getPendingOperations().pop(&send);
    }
}

__device__ void progressAllocatedSend(PendingOperation& send) {
    LOG("progressAllocatedSend() %p", &send);

    volatile SharedThreadState& threadState = sharedState().sharedThreadState[send.otherThread];
    LOG("trying to lock incoming fragments of other thread %d", send.otherThread);
    if (!threadState.fragLock.tryLock()) {
        LOG("fragment lock failed");
        return;
    }
    LOG("fragment lock succeed");
    
    if (threadState.incomingFragments.full()) {
        LOG("incoming fragments list is full, retry later");
        LOG("unlocking list of incoming fragments");
        threadState.fragLock.unlock();
        return;
    }

    IncomingFragment fr;
    assert(send.fragment); // fragment should be allocated
    fr.fragment = send.fragment;
    fr.privatePointer = send.foreignPendingOperation;
    LOG("Pointer to foreign pending operation %p", send.foreignPendingOperation);
    assert(send.foreignPendingOperation);
    assert(fr.privatePointer);
    assert(fr.fragment);

    LOG("put fragment %p into list of incoming fragments", fr.fragment);
    threadState.incomingFragments.push(fr);

    LOG("unlocking list of incoming fragments");
    threadState.fragLock.unlock();

    if (send.count == 0) {
        LOG("All buffer data is already inside fragment, change state to COMPLETED");
        send.state = PendingOperation::State::COMPLETED;
        progressCompletedSend(send);
    } else {
        LOG("Change state to SYNCED (fragment now on the other thread)");
        send.state = PendingOperation::State::SYNCED;
        progressSyncedSend(send);
    }
}

__device__ void progressMatchedSend(PendingOperation& send) {
    LOG("progressMatchedSend() %p", &send);

    LOG("Trying to allocate memory fragment");
    SharedFragmentBuffer& fb = sharedState().sharedFragmentBuffer;
    volatile MemoryFragment* memoryFragment = fb.tryLockFreeFragment();
    if (!memoryFragment) {
        LOG("Memory fragment allocation is failed");
        return;
    }
    LOG("Memory fragment %p allocation is succeed", memoryFragment);

    int copySize = 0;
    void* srcPtr = nullptr;
    LOG("Compare fragment buffer size %d and data size %d", memoryFragment->data.size(), send.count);
    if (memoryFragment->data.size() < send.count) {
        LOG("Fragment buffer size less than data size");
        copySize = memoryFragment->data.size();
        srcPtr = send.data;
        send.data = (void*)(((char*)send.data) + copySize);
        send.count -= copySize;
        LOG("Change state to allocated");
        send.state = PendingOperation::State::ALLOCATED;
    } else {
        LOG("Fragment buffer size greater or equal to data size");
        copySize = send.count;
        srcPtr = send.data;
        send.data = nullptr;
        send.count = 0;
        // we can't mark it as completed because other thread didn't received pointer to fragment
    }
    LOG("Copying data from local memory into memory fragment");
    memcpy_volatile(&memoryFragment->data[0], srcPtr, copySize);

    LOG("Transfer ownership of memory fragment to thread %d", send.otherThread);
    memoryFragment->ownerProcess = send.otherThread;

    LOG("Memory fragment of local pending operation is set to %p", memoryFragment);
    send.fragment = memoryFragment;

    send.state = PendingOperation::State::ALLOCATED;
    progressAllocatedSend(send);
}

__device__ void progressStartedSend(PendingOperation& send, ProgressState& state) {
    LOG("progressStartedSend() %p", &send);
    volatile SharedThreadState& otherThreadState = sharedState().sharedThreadState[send.otherThread];
    
    if (state.isStartedSendSkip(send.otherThread)) {
        LOG("Skip send, because some earlier started send is not processed");
        return;
    }
    
    auto startedSkipGuard = makeScopeGuard([&state,&send](){ 
        state.markStartedSendSkip(send.otherThread); 
    });

    int src = cg::this_grid().thread_rank();

    LOG("Trying to lock state of other process");
    if (!otherThreadState.recvLock.tryLock()) {
        LOG("Failed to lock state of other process");
        return;
    }
    LOG("State of other process is locked");

    volatile auto& uq = otherThreadState.unexpectedRecv;
    volatile auto& rq = otherThreadState.expectedRecv;

    volatile MessageDescriptor* matchedRecv = nullptr;

    LOG("Trying to find matching send in the list of expected receives of other process");
    for (volatile MessageDescriptor* md = rq.head(); md != nullptr; md = rq.next(md)) {
        if (md->src != ANY_SRC && md->src != src) continue;
        if (md->comm != send.comm) continue;
        if (md->tag != ANY_TAG && md->tag != send.tag) continue;
        // if we are here then "md" matches "send"
        matchedRecv = md;
        LOG("Matching receive is found, src: %d (this thread), dst: %d, count: %d", md->src, send.otherThread, send.count);
        break;
    }

    if (matchedRecv) {
        LOG("Remove receive from the list of expected receives of other process");
        send.foreignPendingOperation = matchedRecv->privatePointer;
        assert(send.foreignPendingOperation);
        LOG("Pointer to foregin pending operation is %p", send.foreignPendingOperation);
        rq.pop(matchedRecv);
        LOG("Change state to MATCHED");
        send.state = PendingOperation::State::MATCHED;
    } else {
        LOG("Matching receive is not found, post send in unexpected receives of other process");

        if (uq.full()) {
            LOG("List of unexpected receives is full, retry later");
        } else {
            MessageDescriptor md;
            md.comm = send.comm;
            md.src = src;
            md.tag = send.tag;
            md.privatePointer = &send;
            uq.push(md);
            LOG("Change state to POSTED");
            send.state = PendingOperation::State::POSTED;
        }
    }
    
    LOG("Unlock state of other process");
    otherThreadState.recvLock.unlock();

    if (send.state != PendingOperation::State::STARTED) {
        startedSkipGuard.commit();
    }
    
    if (send.state == PendingOperation::State::MATCHED) {
        progressMatchedSend(send);
    } else if (send.state == PendingOperation::State::POSTED) {
        progressPostedSend(send);
    }
}

__device__ void progressPostedSend(PendingOperation& send) {
    LOG("progressPostedSend() %p", &send);

    if (send.fragment != nullptr) {
        LOG("Fragment is allocated by other thread, change state to SYNCED");
        send.state = PendingOperation::State::SYNCED;
        progressSyncedSend(send);
    } else {
        LOG("Fragment is not allocated by other thread, skip it");
    }
}

__device__ void progressSyncedSend(PendingOperation& send) {
    LOG("progressSyncedSend() %p", &send);

    LOG("check the owner of shared fragment buffer");
    if (send.fragment->ownerProcess == send.otherThread) {
        LOG("buffer is owned by other thread, skip it");
        return;
    }
    LOG("buffer is owned by me, continue operation");

    int copySize = 0;
    void* srcPtr = nullptr;
    LOG("Send %p, fragment size %d, data to be sent %d",
        &send,
        send.fragment->data.size(),
        send.count
    );
    if (send.fragment->data.size() < send.count) {
        LOG("copy next chunk, it is not the last one");
        // a lot of chunks left
        copySize = send.fragment->data.size();
        srcPtr = send.data;
        send.data = (void*)((char*)send.data + copySize);
        send.count -= copySize;
    } else {
        LOG("it is last chunk");
        // last chunk
        copySize = send.count;
        srcPtr = send.data;
        send.data = nullptr;
        send.count = 0;
        LOG("copy last chunk, change state to COMPLETED");
        send.state = PendingOperation::State::COMPLETED;
    }
    LOG("copy chunk from local buffer to destionation buffer");
    memcpy_volatile(&send.fragment->data[0], srcPtr, copySize);

    LOG("transfer ownership of shared fragment %p to other thread", send.fragment);
    send.fragment->ownerProcess = send.otherThread;

    if (send.state == PendingOperation::State::COMPLETED) {
        progressCompletedSend(send);
    }
}

__device__ void progressSend(PendingOperation& send, ProgressState& state) {
    LOG("progressSend() %p", &send);

    switch (send.state) {
        case PendingOperation::State::STARTED:
            progressStartedSend(send, state);
            break;
        case PendingOperation::State::POSTED:
            progressPostedSend(send);
            break;
        case PendingOperation::State::MATCHED:
            progressMatchedSend(send);
            break;
        case PendingOperation::State::ALLOCATED:
            progressAllocatedSend(send);
            break;
        case PendingOperation::State::SYNCED:
            progressSyncedSend(send);
            break;
        case PendingOperation::State::COMPLETED:
            progressCompletedSend(send);
            break;
    }
}

__device__ void progressStartedRecv(PendingOperation& recv, ProgressState& state) {
    LOG("progressStartedRecv() %p", &recv);

    int dst = cg::this_grid().thread_rank();
    
    if (state.isStartedRecvSkip(recv.otherThread)) {
        LOG("Skip recv, because some earlier started recv is not processed");
        return;
    }
    
    auto startedSkipGuard = makeScopeGuard([&state,&recv](){ 
        state.markStartedRecvSkip(recv.otherThread); 
    });

    volatile SharedThreadState& currentThreadState = sharedState().sharedThreadState[dst];

    LOG("Trying to take lock for shared thread state of current thread");
    if (!currentThreadState.recvLock.tryLock()) {
        LOG("Failed to take lock");
        return;
    }
    LOG("Lock is taken successfully");

    volatile auto& uq = currentThreadState.unexpectedRecv;
    volatile auto& rq = currentThreadState.expectedRecv;

    volatile MessageDescriptor* matchedSend = nullptr;

    LOG("Trying to find message in the list of unexpected messages");
    for (volatile MessageDescriptor* md = uq.head(); md != nullptr; md = uq.next(md)) {
        if (md->src != recv.otherThread) continue;
        if (md->comm != recv.comm) continue;
        if (md->tag != recv.tag) continue;
        // if we are here then "md" matches "recv"
        LOG("Message is found in unexpected list");
        matchedSend = md;
        break;
    }

    if (matchedSend) {
        LOG("Save pointer to `send` operation of other process");
        recv.foreignPendingOperation = matchedSend->privatePointer;
        LOG("Remove message from list of unexpected messages");
        uq.pop(matchedSend);

        LOG("Change state to MATCHED");
        recv.state = PendingOperation::State::MATCHED;
    } else {
        LOG("Add message to the list of expected receives of current threads");
        
        if (rq.full()) {
            LOG("List of expected receives is full, retry later");
        } else {
            MessageDescriptor md;
            md.comm = recv.comm;
            md.src = recv.otherThread;
            md.tag = recv.tag;
            md.privatePointer = &recv;
            rq.push(md);

            LOG("Change state to POSTED");
            recv.state = PendingOperation::State::POSTED;
        }
    }

    LOG("Unlock shared state of current thread");
    currentThreadState.recvLock.unlock();

    if (recv.state != PendingOperation::State::STARTED) {
        startedSkipGuard.commit();
    }
    
    if (recv.state == PendingOperation::State::MATCHED) {
        progressMatchedRecv(recv);
    } else if (recv.state == PendingOperation::State::POSTED) {
        progressPostedRecv(recv);
    }
}


__device__ void progressPostedRecv(PendingOperation& recv) {
    LOG("progressPostedRecv() %p", &recv);

    if (recv.fragment != nullptr) {
        LOG("Fragment is allocated by other thread, change state to SYNCED");
        recv.state = PendingOperation::State::SYNCED;
        progressSyncedRecv(recv);
    } else {
        LOG("Fragment is not allocated by other thread, skip it");
    }
}

__device__ void progressMatchedRecv(PendingOperation& recv) {
    LOG("progressMatchedRecv() %p", &recv);

    LOG("Trying lock free memory fragment");
    SharedFragmentBuffer& fb = sharedState().sharedFragmentBuffer;
    volatile MemoryFragment* memoryFragment = fb.tryLockFreeFragment();
    if (!memoryFragment) {
        LOG("Failed to lock memory fragment");
        return;
    }
    LOG("Memory fragment %p is locked", memoryFragment);

    LOG("Transfer ownership of fragment to other thread");
    memoryFragment->ownerProcess = recv.otherThread;

    LOG("Memory fragment %p, owner %d", memoryFragment, memoryFragment->ownerProcess);
    
    recv.fragment = memoryFragment;

    LOG("Change state to ALLOCATED");
    recv.state = PendingOperation::State::ALLOCATED;

    progressAllocatedRecv(recv);
}

__device__ void progressAllocatedRecv(PendingOperation& recv) {
    LOG("progressAllocatedRecv() %p", &recv);

    LOG("Trying to lock list of incoming fragments of thread %d", recv.otherThread);
    volatile SharedThreadState& threadState = sharedState().sharedThreadState[recv.otherThread];
    if (!threadState.fragLock.tryLock()) {
        LOG("Failed to lock");
        return;
    }
    LOG("Locked successfully");
    
    if (threadState.incomingFragments.full()) {
        LOG("incoming fragments list is full, retry later");
        LOG("unlocking list of incoming fragments");
        threadState.fragLock.unlock();
        return;
    }

    LOG("RECEIVE: %p, FRAGMENT: %p", &recv, recv.fragment);
    
    LOG("Memory fragment %p, owner %d", recv.fragment, recv.fragment->ownerProcess);

    IncomingFragment fr;
    fr.fragment = recv.fragment;
    fr.privatePointer = recv.foreignPendingOperation;

    assert(fr.fragment);
    assert(fr.privatePointer);

    LOG("Put new fragment %p (operation %p) into list of incoming fragments",
        fr.fragment, fr.privatePointer
    );
    threadState.incomingFragments.push(fr);

    LOG("Unlock list of incoming fragments of other thread %d", recv.otherThread);
    threadState.fragLock.unlock();

    LOG("Change state to SYNCED");
    recv.state = PendingOperation::State::SYNCED;

    LOG("RECEIVE: %p, FRAGMENT: %p", &recv, recv.fragment);
    
    progressSyncedRecv(recv);
}

__device__ void progressSyncedRecv(PendingOperation& recv) {
    LOG("progressSyncedRecv() %p", &recv);
    
    LOG("RECEIVE: %p, FRAGMENT: %p", &recv, recv.fragment);

    LOG("Memory fragment %p, owner %d", recv.fragment, recv.fragment->ownerProcess);
    
    LOG("Check that current thread owns fragment");
    if (recv.fragment->ownerProcess == recv.otherThread) {
        LOG("Fragment is used by other process, skip it");
        return;
    }
    LOG("Fragment %p is owned by current thread", recv.fragment);

    int copySize = 0;
    void* dstPtr = nullptr;
    LOG("Receive %p, fragment size %d, data to be received %d",
        &recv,
        recv.fragment->data.size(),
        recv.count
    );
    if (recv.fragment->data.size() < recv.count) {
        LOG("Prepare copy of next chunk");
        // a lot of chunks left
        copySize = recv.fragment->data.size();
        dstPtr = recv.data;
        recv.data = (void*)((char*)recv.data + copySize);
        recv.count -= copySize;
    } else {
        LOG("Prepare copy of last chunk");
        // last chunk
        copySize = recv.count;
        dstPtr = recv.data;
        recv.data = nullptr;
        recv.count = 0;
        LOG("Change state to COMPLETED");
        recv.state = PendingOperation::State::COMPLETED;
    }
    LOG("Copy data from fragment buffer into local memory");
    memcpy_volatile(dstPtr, &recv.fragment->data[0], copySize);

    if (recv.state == PendingOperation::State::COMPLETED) {
        progressCompletedRecv(recv);
    } else if (recv.state == PendingOperation::State::SYNCED) {
        LOG("Transfer fragment ownership to other thread");
        recv.fragment->ownerProcess = recv.otherThread;
    } else {
        assert(0);
    }
}

__device__ void progressRecv(PendingOperation& recv, ProgressState& state) {
    LOG("progressRecv() %p", &recv);

    switch (recv.state) {
        case PendingOperation::State::STARTED:
            progressStartedRecv(recv, state);
            break;
        case PendingOperation::State::POSTED:
            progressPostedRecv(recv);
            break;
        case PendingOperation::State::MATCHED:
            progressMatchedRecv(recv);
            break;
        case PendingOperation::State::ALLOCATED:
            progressAllocatedRecv(recv);
            break;
        case PendingOperation::State::SYNCED:
            progressSyncedRecv(recv);
            break;
        case PendingOperation::State::COMPLETED:
            progressCompletedRecv(recv);
            break;
    }
}

__device__ void receiveFragmentPointers() {
    LOG("receiveFragmentPointers()");

    int curThread = cg::this_grid().thread_rank();
    SharedState& ss = sharedState();
    volatile SharedThreadState& sts = ss.sharedThreadState[curThread];

    LOG("Trying to lock list of incoming fragment of current thread");
    if (!sts.fragLock.tryLock()) {
        LOG("Failed to lock");
        return;
    }
    LOG("Locked successfully");

    LOG("Looping over incoming fragments");
    while (!sts.incomingFragments.empty()) {
        volatile IncomingFragment* inFrag = sts.incomingFragments.head();
        assert(inFrag);
        LOG("Found incoming fragment at address %p", inFrag);

        PendingOperation* pop = inFrag->privatePointer;
        LOG("Extract pointer to private pending operation %p", pop);
        assert(pop);
        
        volatile MemoryFragment* frag = inFrag->fragment;
        assert(frag);

        assert(!pop->fragment);

        LOG("Assign incoming fragment %p to the private pending operation %p", frag, pop);
        pop->fragment = frag;

        LOG("Remove fragment from the list of incoming fragments");
        sts.incomingFragments.pop(inFrag);
    }

    LOG("Unlock list of incoming fragments of current thread");
    sts.fragLock.unlock();
}

__host__ __device__ void progress() {
    // this function is no op on host
#if defined(__CUDA_ARCH__)
    LOG("progress()");

    receiveFragmentPointers();

    ProgressState progressState;
    
    auto& pops = threadPrivateState().getPendingOperations();
    for (volatile PendingOperation* ptr = pops.head(); ptr != nullptr; ptr = pops.next(ptr)) {
        PendingOperation& pop = *(PendingOperation*)ptr; // TODO: remove ugly cast
        switch (pop.type) {
            case PendingOperation::Type::SEND:
                progressSend(pop, progressState);
                break;
            case PendingOperation::Type::RECV:
                progressRecv(pop, progressState);
                break;
        }
    }
#endif
}

__device__ bool test(PendingOperation* op) {
    LOG("test()");
    assert(op->canBeFreed == false);
    progress();
    if (op->state == PendingOperation::State::COMPLETED) {
        op->canBeFreed = true;
        switch (op->type) {
            case PendingOperation::Type::SEND:
                progressCompletedSend(*op);
                break;
            case PendingOperation::Type::RECV:
                progressCompletedRecv(*op);
                break;
        }
        return true;
    }
    return false;
}

__device__ void wait(PendingOperation* op) {
    LOG("wait()");
    assert(op->canBeFreed == false);
    while (!test(op)) {}
}


DeviceToHostCommunicator::DeviceToHostCommunicator(size_t queueSize, size_t numThreads)
    : queue(queueSize)
    , hostFinished(numThreads, false)
{
}

__device__ void DeviceToHostCommunicator::delegateToHost(void* ptr, size_t size) {
    int threadRank = cg::this_multi_grid().thread_rank();
    assert(hostFinished[threadRank] == false);

    while (true) {

        while (!lock.tryLock()) {
            progress();
        }

        auto unlockGuard = makeScopeGuard([&](){ lock.unlock(); });

        if (queue.full()) {
            lock.unlock();
            unlockGuard.commit();

            progress();
        } else {
            queue.push(Message(ptr, size, threadRank));

            break;
        }
    }

    // waiting for host
    while (!hostFinished[threadRank]) {
        progress();
    }

    hostFinished[threadRank] = false;
}


FreeManagedMemory::FreeManagedMemory(size_t size)
    : buffer(size)
{
    assert(buffer.size() > sizeof(BlockDescriptor));
    BlockDescriptor* memBlock = (BlockDescriptor*)(&buffer[0]);
    memBlock->status = FREE;
    memBlock->end = buffer.size();
}

__host__ __device__ void* FreeManagedMemory::allocate(size_t size) {
    while (!lock.tryLock()) {
        progress();
    }

    auto unlockGuard = makeScopeGuard([&](){ lock.unlock(); });

    size_t blockStart = 0;
    while (true) {
        BlockDescriptor* memBlock = (BlockDescriptor*)(&buffer[blockStart]);
        size_t blockDataStart = blockStart + sizeof(BlockDescriptor);
        assert(memBlock->end > blockStart);
        size_t blockUsefulSize = memBlock->end - blockStart;

        compactionWithNextBlocks(blockStart);

        if (memBlock->status == FREE && blockUsefulSize >= size) {
            // allocate block
            size_t blockSizeLeft = blockUsefulSize - size;
            if (blockSizeLeft <= sizeof(BlockDescriptor)) {
                // utilize all memory since it will not be possibe to use it anyway
                memBlock->status = USED;
                return (void*) &buffer[blockDataStart];
            } else {
                // normal allocation, split buffer into two parts: first is allocated, the second is free
                size_t newBlockEnd = blockDataStart + size;

                // second free block
                BlockDescriptor* newFreeBlock = (BlockDescriptor*)(&buffer[newBlockEnd]);
                newFreeBlock->status = FREE;
                newFreeBlock->end = memBlock->end;

                // first used block
                memBlock->status = USED;
                memBlock->end = newBlockEnd;
            }
        }

        blockStart = memBlock->end;

        assert(blockStart <= buffer.size());

        if (blockStart == buffer.size()) {
            return nullptr;
        }
    }
}

__host__ __device__ void FreeManagedMemory::free(void* ptr) {
    while (!lock.tryLock()) {
        progress();
    }

    auto unlockGuard = makeScopeGuard([&](){ lock.unlock(); });

    assert(&buffer[0] <= ptr);
    size_t pos = ((char*)ptr) - &buffer[0];
    assert(pos < buffer.size());

    BlockDescriptor* memBlock = (BlockDescriptor*)(&ptr);
    assert(memBlock->status == USED);

    memBlock->status = FREE;
}

__host__ __device__ void FreeManagedMemory::compactionWithNextBlocks(size_t currentBlock) {
    BlockDescriptor* current = (BlockDescriptor*)(&buffer[currentBlock]);

    while (true) {
        size_t nextBlock = current->end;
        assert(nextBlock <= buffer.size());

        if (nextBlock == buffer.size()) break;

        BlockDescriptor* next = (BlockDescriptor*)(&buffer[nextBlock]);
        if (next->status == USED) break;

        assert(next->status == FREE);

        current->end = next->end;
    }
}

} // namespace
