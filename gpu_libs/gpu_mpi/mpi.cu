#include "hip/hip_runtime.h"
#include "datatypes.cuh"
#include "mpi.cuh"

// cuda_mpi.cuh should be included before device specific standard library functions
// because it relies on standard ones
#include "cuda_mpi.cuh"

#include "stdlib.cuh"
#include "string.cuh"

#include "mpi_common.cuh"

#include "device_vector.cuh"

#include "operators.cuh"

#define MPI_COLLECTIVE_TAG (-2)

//nio
#define USE_AIO true
#define USE_URING false

// internal opaque object
struct MPI_Request_impl {
    //TODO: need better implementation, should not direct IOOperation in the declairation of the func.

    
    enum Type { SR, IO };

    union {
        CudaMPI::PendingOperation* pendingOperation;
        CudaMPI::PendingIOOperation* pendingIOOperation;
    };

    Type type;
    int ref_count;

    __device__ MPI_Request_impl(CudaMPI::PendingOperation* pendingOperation) 
        : ref_count(1) 
        , pendingOperation(pendingOperation) 
    {this->type = SR; }
    __device__ MPI_Request_impl(CudaMPI::PendingIOOperation* ioop, int io) 
        : ref_count(1)
        , pendingIOOperation(ioop)
    {this->type = Type(io);}
};
namespace gpu_mpi {
    
__device__ void incRequestRefCount(MPI_Request request) {
    assert(request->ref_count > 0);
    request->ref_count++;
}

#undef MPI_TYPES_LIST

} // namespace

__device__ int MPI_Init(int *argc, char ***argv) {
    gpu_mpi::initializeGlobalGroups();
    gpu_mpi::initializeGlobalCommunicators();
    gpu_mpi::initializeOps();
    return MPI_SUCCESS;
}

__device__ int MPI_Init_thread(int *argc, char ***argv, int required, int *provided) {
    (void) required;
    *provided = MPI_THREAD_SINGLE;
    return MPI_Init(argc, argv);
}

__device__ int MPI_Finalize(void) {
    // TODO: due to exit() you need to perform
    // all MPI related memory deallocation here


    gpu_mpi::destroyGlobalGroups();
    gpu_mpi::destroyGlobalCommunicators();
    
    gpu_mpi::destroyOps();
    
    return MPI_SUCCESS;
}

__device__ int MPI_Get_processor_name(char *name, int *resultlen) {
    const char hardcoded_name[] = "GPU thread";
    __gpu_strcpy(name, hardcoded_name);
    *resultlen = sizeof(hardcoded_name);
    return MPI_SUCCESS;
}

static __device__ CudaMPI::DeviceVector<char>* native_buf;
__device__ CudaMPI::DeviceVector<char>& nativeBuf() {
    if (!native_buf) {
        native_buf = new CudaMPI::DeviceVector<char>;
    }
    __gpu_assert(native_buf);
    return *native_buf;
}

__device__ int MPI_Bcast_native(void* buffer, int size, int root) {
    int commRank = -1;
    MPI_Comm_rank(MPI_COMM_WORLD, &commRank);
    CudaMPI::sharedState().gridBarrier();
    if (root == commRank) {
        nativeBuf().resize(size);
        __gpu_memcpy(&nativeBuf()[0], buffer, size);
    }
    CudaMPI::sharedState().gridBarrier();
    if (root != commRank) {
        __gpu_memcpy(buffer, &nativeBuf()[0], size);
    }
    return MPI_SUCCESS;
}

__device__ int MPI_Bcast(void *buffer, int count, MPI_Datatype datatype,
                         int root, MPI_Comm comm)
{
    int dataSize = gpu_mpi::TypeSize(datatype) * count;
    assert(dataSize > 0);

    if (comm == MPI_COMM_WORLD) {
        return MPI_Bcast_native(buffer, dataSize, root);
    }
    
    int commSize = -1;
    int commRank = -1;
    
    MPI_Comm_size(comm, &commSize);
    MPI_Comm_rank(comm, &commRank);
    
    int tag = MPI_COLLECTIVE_TAG;
    int ctx = gpu_mpi::getCommContext(comm);
    
    if (commRank == root) {
        CudaMPI::PendingOperation** ops = (CudaMPI::PendingOperation**) malloc(sizeof(CudaMPI::PendingOperation*) * commSize);
        assert(ops);
        for (int dst = 0; dst < commSize; dst++) {
            if (dst != commRank) {
                ops[dst] = CudaMPI::isend(dst, buffer, dataSize, ctx, tag);
            }
        }
        for (int dst = 0; dst < commSize; dst++) {
            if (dst != commRank) {
                CudaMPI::wait(ops[dst]);
            }
        }
        free(ops);
    } else {
        CudaMPI::PendingOperation* op = CudaMPI::irecv(root, buffer, dataSize, ctx, tag);
        CudaMPI::wait(op);
    }
    
    return MPI_SUCCESS;
}

__device__ double MPI_Wtime(void) {
    auto clock = clock64();
    double seconds = clock * MPI_Wtick();
    return seconds;
}

__device__ int MPI_Reduce_native(
    const void *sendbuf, void *recvbuf, int count, int root) 
{
    int commRank = -1;
    MPI_Comm_rank(MPI_COMM_WORLD, &commRank);

    int elemSize = gpu_mpi::plainTypeSize(MPI_DOUBLE);
    int dataSize = elemSize * count;
    __gpu_assert(dataSize > 0);
    CudaMPI::sharedState().gridBarrier();
    if (root == commRank) {
        nativeBuf().resize(dataSize);
        double* native_buf_start = (double*)&nativeBuf()[0];
        for (int i = 0; i < count; i++) {
            native_buf_start[i] = ((double*)sendbuf)[i];
        }
    }
    CudaMPI::sharedState().gridBarrier();
    double* native_buf_start = (double*)&nativeBuf()[0];
    if (root != commRank) {
        for (int i = 0; i < count; i++) {
            atomicAdd(&native_buf_start[i], ((double*)sendbuf)[i]);
        }
    }
    CudaMPI::sharedState().gridBarrier();
    if (root == commRank) {
        for (int i = 0; i < count; i++) {
            ((double*)recvbuf)[i] = native_buf_start[i];
        }
    }
    return MPI_SUCCESS;
}

__device__ int MPI_Reduce(const void *sendbuf, void *recvbuf, int count,
                          MPI_Datatype datatype, MPI_Op op, int root, MPI_Comm comm)
{
    if (comm == MPI_COMM_WORLD && op == MPI_SUM && datatype == MPI_DOUBLE) {
        return MPI_Reduce_native(sendbuf, recvbuf, count, root);
    }

    int elemSize = gpu_mpi::TypeSize(datatype);
    int dataSize = elemSize * count;
    __gpu_assert(dataSize > 0);
    
    int commSize = -1;
    int commRank = -1;
    MPI_Comm_size(comm, &commSize);
    MPI_Comm_rank(comm, &commRank);
    
    int tag = MPI_COLLECTIVE_TAG;
    int ctx = gpu_mpi::getCommContext(comm);
    
    if (commRank == root) {
        auto ops = (CudaMPI::PendingOperation**) malloc(sizeof(CudaMPI::PendingOperation*) * commSize);
        void* buffers = malloc(dataSize * commSize);
        assert(ops);
        for (int src = 0; src < commSize; src++) {
            if (src != commRank) {
                ops[src] = CudaMPI::irecv(src, ((char*)buffers) + src * dataSize, dataSize, ctx, tag);
            }
        }
        for (int src = 0; src < commSize; src++) {
            const void* tempbuf = nullptr;
            if (src != commRank) {
                CudaMPI::wait(ops[src]);
                tempbuf = ((char*)buffers) + src * dataSize;
            } else {
                tempbuf = sendbuf;
            }
            
            if (src == 0) {
                for (int i = 0; i < dataSize; i++) {
                    ((char*)recvbuf)[i] = ((char*)tempbuf)[i];
                }
            } else {
                gpu_mpi::invokeOperator(op, tempbuf, recvbuf, &count, &datatype);
            }
        }
        
        free(buffers);
        free(ops);
    } else {
        CudaMPI::PendingOperation* op = CudaMPI::isend(root, sendbuf, dataSize, ctx, tag);
        CudaMPI::wait(op);
    }
    
    return MPI_SUCCESS;
}

// extern std::list<MPI_Datatype*> Typelist;

__device__ int MPI_Type_contiguous(int count, MPI_Datatype oldtype, MPI_Datatype *newtype) {
    // newtype -> _size = count * oldtype.size();
    // newtype -> typemap_len = count * oldtype.typemap_len;
    // for (int i = 0; i < count; i++) copy_typemap_once(newtype, oldtype, i, 0);
    // newtype -> committed = false;
    MPI_Datatype contiguousType;
    for (int i = 0; i < count; i++) contiguousType.add_typemap_at_end(oldtype, 0);
    *newtype = contiguousType;
    return MPI_SUCCESS;
}

__device__ int MPI_Type_vector(int count, int blocklength, int stride, MPI_Datatype oldtype, MPI_Datatype *newtype) {
    if (blocklength == 0 || stride == 0) return MPI_ERR_OTHER;
    if (stride < blocklength) return MPI_ERR_OTHER;
    // In all our use cases, we do not allow a typemap to overlap itself.

    MPI_Datatype vectorType;
    int gap = (int)(stride - blocklength) * oldtype.size();
    for(int i = 0; i < count; i++){
        vectorType.add_typemap_at_end(oldtype, gap);
        for(int j = 1; j < blocklength; j++){
            vectorType.add_typemap_at_end(oldtype, 0);
        }
    }
    vectorType.typemap_gap = gap;

    *newtype = vectorType;  
    return MPI_SUCCESS;
}

__device__ int MPI_Type_commit(MPI_Datatype *datatype) {
    if (datatype -> committed == true) return MPI_SUCCESS;
    datatype -> committed = true;  // Typelist.push_back(datatype);
    return MPI_SUCCESS;
}

__device__ int MPI_Recv(void *buf, int count, MPI_Datatype datatype,
                        int source, int tag, MPI_Comm comm, MPI_Status *status) {
    MPI_Request request;
    MPI_Irecv(buf, count, datatype, source, tag, comm, &request);
    MPI_Wait(&request, MPI_STATUS_IGNORE);
    return MPI_SUCCESS;
}

__device__ int MPI_Sendrecv(const void *sendbuf, int sendcount, MPI_Datatype sendtype,
            int dest, int sendtag, void *recvbuf, int recvcount,
            MPI_Datatype recvtype, int source, int recvtag,
                 MPI_Comm comm, MPI_Status *status) {
    return MPI_SUCCESS;
}

__device__ int MPI_Send(const void *buf, int count, MPI_Datatype datatype, int dest,
            int tag, MPI_Comm comm)
{
    MPI_Request request;
    MPI_Isend(buf, count, datatype, dest, tag, comm, &request);
    MPI_Wait(&request, MPI_STATUS_IGNORE);
    return MPI_SUCCESS;
}

__device__ double MPI_Wtick() {
    int peakClockKHz = CudaMPI::threadPrivateState().peakClockKHz;
    return 0.001 / peakClockKHz;
}

__device__ int MPI_Allreduce(const void *sendbuf, void *recvbuf, int count,
                         MPI_Datatype datatype, MPI_Op op, MPI_Comm comm)
{
    int err = MPI_Reduce(sendbuf, recvbuf, count, datatype, op, 0, comm);
    if (err != MPI_SUCCESS) return err;
    return MPI_Bcast(recvbuf, count, datatype, 0, comm);
}
__device__ int MPI_Abort(MPI_Comm comm, int errorcode) {
    NOT_IMPLEMENTED;
    return MPI_SUCCESS;
}
__device__ int MPI_Type_size(MPI_Datatype datatype, int *size) {
    NOT_IMPLEMENTED;
    return MPI_SUCCESS;
}
__device__ int MPI_Gather(const void *sendbuf, int sendcount, MPI_Datatype sendtype,
                          void *recvbuf, int recvcount, MPI_Datatype recvtype, int root,
                          MPI_Comm comm)
{
    // TODO implement through MPI_Gatherv
    int comm_size = -1;
    int comm_rank = -1;
    MPI_Comm_size(comm, &comm_size);
    MPI_Comm_rank(comm, &comm_rank);

    int sendElemSize = gpu_mpi::TypeSize(sendtype);
    int recvElemSize = gpu_mpi::TypeSize(recvtype);
    assert(sendElemSize > 0);
    assert(recvElemSize > 0);

    assert(sendElemSize * sendcount == recvElemSize * recvcount);
    int dataSize = sendElemSize * sendcount;

    if (comm_rank != root) {
        MPI_Send(sendbuf, sendcount, sendtype, root, MPI_COLLECTIVE_TAG, comm);
    } else {
        for (int r = 0; r < comm_size; r++) {
            if (r == root) {
                memcpy(((char*)recvbuf) + r * dataSize, sendbuf, dataSize);
            } else {
                MPI_Recv(((char*)recvbuf) + r * dataSize, recvcount, recvtype, r, MPI_COLLECTIVE_TAG, comm, MPI_STATUS_IGNORE);
            }
        }
    }
    
    return MPI_SUCCESS;
}

__device__ int MPI_Barrier(MPI_Comm comm) {
    if (comm == MPI_COMM_WORLD) { 
        CudaMPI::sharedState().gridBarrier();
    } else {
        NOT_IMPLEMENTED;
    }
    return MPI_SUCCESS;
}

__device__ int MPI_Alltoall(
    const void *sendbuf, int sendcount, MPI_Datatype sendtype,
    void *recvbuf, int recvcount, MPI_Datatype recvtype,
    MPI_Comm comm)
{
    int comm_size = -1;
    MPI_Comm_size(comm, &comm_size);

    int* sdispls = (int*) malloc(comm_size * sizeof(int));
    int* rdispls = (int*) malloc(comm_size * sizeof(int));
    int* sendcounts = (int*) malloc(comm_size * sizeof(int));
    int* recvcounts = (int*) malloc(comm_size * sizeof(int));

    for (int i = 0; i < comm_size; i++) {
        sdispls[i] = i * sendcount;
        rdispls[i] = i * recvcount;
        sendcounts[i] = sendcount;
        recvcounts[i] = recvcount;
    }
    int res = MPI_Alltoallv(
        sendbuf, sendcounts, sdispls, sendtype,
        recvbuf, recvcounts, rdispls, recvtype, comm);

    free(sdispls);
    free(rdispls);
    free(sendcounts);
    free(recvcounts);

    return res;
}
__device__ int MPI_Alltoallv(
    const void *sendbuf, const int sendcounts[], const int sdispls[], MPI_Datatype sendtype,
    void *recvbuf, const int recvcounts[], const int rdispls[], MPI_Datatype recvtype, 
    MPI_Comm comm) 
{
    int comm_size = -1;
    int comm_rank = -1;
    MPI_Comm_size(comm, &comm_size);
    MPI_Comm_rank(comm, &comm_rank);

    int sendElemSize = gpu_mpi::TypeSize(sendtype);
    int recvElemSize = gpu_mpi::TypeSize(recvtype);

    MPI_Request* send_requests = (MPI_Request*) malloc(sizeof(MPI_Request) * comm_size);
    MPI_Request* recv_requests = (MPI_Request*) malloc(sizeof(MPI_Request) * comm_size);
    assert(send_requests && "Can't allocate memory");
    assert(recv_requests && "Can't allocate memory");

    for (int i = 0; i < comm_size; i++) {
        if (i != comm_rank) {
            MPI_Isend(((char*)sendbuf) + sdispls[i] * sendElemSize, sendcounts[i], sendtype, i, MPI_COLLECTIVE_TAG, comm, &send_requests[i]);
        }
    }

    for (int i = 0; i < comm_size; i++) {
        if (i != comm_rank) {
            MPI_Irecv(((char*)recvbuf) + rdispls[i] * recvElemSize, recvcounts[i], recvtype, i, MPI_COLLECTIVE_TAG, comm, &recv_requests[i]);
        }
    }

    memcpy(((char*)recvbuf) + rdispls[comm_rank] * recvElemSize, 
           ((char*)sendbuf) + sdispls[comm_rank] * sendElemSize,
           recvcounts[comm_rank] * recvElemSize);

    for (int i = 0; i < comm_size; i++) {
        if (i != comm_rank) {
            MPI_Wait(&send_requests[i], MPI_STATUS_IGNORE);
            MPI_Wait(&recv_requests[i], MPI_STATUS_IGNORE);
        }
    }

    free(send_requests);
    free(recv_requests);

    return MPI_SUCCESS;
}

__device__ int MPI_Allgather(const void *sendbuf, int  sendcount,
             MPI_Datatype sendtype, void *recvbuf, int recvcount,
             MPI_Datatype recvtype, MPI_Comm comm)
{
    MPI_Gather(sendbuf, sendcount, sendtype, recvbuf, recvcount, recvtype, 0, comm);
    int comm_size = -1;
    MPI_Comm_size(comm, &comm_size);
    MPI_Bcast(recvbuf, recvcount * comm_size, recvtype, 0, comm);
    return MPI_SUCCESS;
}

__device__ int MPI_Allgatherv(const void *sendbuf, int sendcount,
                              MPI_Datatype sendtype, void *recvbuf, const int recvcounts[],
                              const int displs[], MPI_Datatype recvtype, MPI_Comm comm)
{
    NOT_IMPLEMENTED;
    return MPI_SUCCESS;
}

__device__ int MPI_Gatherv(const void *sendbuf, int sendcount, MPI_Datatype sendtype,
                           void *recvbuf, const int recvcounts[], const int displs[], MPI_Datatype recvtype,
                           int root, MPI_Comm comm)
{
    // TODO implement through MPI_Igatherv
    int comm_size = -1;
    int comm_rank = -1;
    MPI_Comm_size(comm, &comm_size);
    MPI_Comm_rank(comm, &comm_rank);

    int sendElemSize = gpu_mpi::TypeSize(sendtype);
    assert(sendElemSize > 0);

    if (comm_rank != root) {
        MPI_Send(sendbuf, sendcount, sendtype, root, MPI_COLLECTIVE_TAG, comm);
    } else {
        int recvElemSize = gpu_mpi::TypeSize(recvtype);
        assert(recvElemSize > 0);
        for (int r = 0; r < comm_size; r++) {
            if (r == root) {
                memcpy(((char*)recvbuf) + displs[r] * recvElemSize, sendbuf, recvcounts[r] * recvElemSize);
            } else {
                MPI_Recv(((char*)recvbuf) + displs[r] * recvElemSize, recvcounts[r], recvtype, r, MPI_COLLECTIVE_TAG, comm, MPI_STATUS_IGNORE);
            }
        }
    }
    
    return MPI_SUCCESS;
}
__device__ int MPI_Scatter(const void *sendbuf, int sendcount, MPI_Datatype sendtype,
                           void *recvbuf, int recvcount, MPI_Datatype recvtype, int root,
                           MPI_Comm comm)
{
    // TODO implement through MPI_Scatterv
    int comm_size = -1;
    int comm_rank = -1;
    MPI_Comm_size(comm, &comm_size);
    MPI_Comm_rank(comm, &comm_rank);

    int sendElemSize = gpu_mpi::TypeSize(sendtype);
    int recvElemSize = gpu_mpi::TypeSize(recvtype);
    assert(sendElemSize > 0);
    assert(recvElemSize > 0);

    assert(sendElemSize * sendcount == recvElemSize * recvcount);
    int dataSize = sendElemSize * sendcount;

    if (comm_rank != root) {
        MPI_Recv(recvbuf, recvcount, recvtype, root, MPI_COLLECTIVE_TAG, comm, MPI_STATUS_IGNORE);
    } else {
        for (int r = 0; r < comm_size; r++) {
            if (r == root) {
                memcpy(recvbuf, ((char*)sendbuf) + r * dataSize, dataSize);
            } else {
                MPI_Send(((char*)sendbuf) + r * dataSize, sendcount, sendtype, r, MPI_COLLECTIVE_TAG, comm);
            }
        }
    }
    
    return MPI_SUCCESS;
}
__device__ int MPI_Scatterv(const void *sendbuf, const int sendcounts[], const int displs[],
                            MPI_Datatype sendtype, void *recvbuf, int recvcount,
                            MPI_Datatype recvtype, int root, MPI_Comm comm)
{
    int comm_size = -1;
    int comm_rank = -1;
    MPI_Comm_size(comm, &comm_size);
    MPI_Comm_rank(comm, &comm_rank);

    int recvElemSize = gpu_mpi::TypeSize(recvtype);
    assert(recvElemSize > 0);

    if (comm_rank != root) {
        MPI_Recv(recvbuf, recvcount, recvtype, root, MPI_COLLECTIVE_TAG, comm, MPI_STATUS_IGNORE);
    } else {
        int sendElemSize = gpu_mpi::TypeSize(sendtype);
        assert(sendElemSize > 0);
        for (int r = 0; r < comm_size; r++) {
            if (r == root) {
                memcpy(recvbuf, ((char*)sendbuf) + displs[r] * sendElemSize, sendcounts[r] * sendElemSize);
            } else {
                MPI_Send(((char*)sendbuf) + displs[r] * sendElemSize, sendcounts[r], sendtype, r, MPI_COLLECTIVE_TAG, comm);
            }
        }
    }
    
    return MPI_SUCCESS;
}

__device__ int MPI_NULL_COPY_FN(MPI_Comm oldcomm, int keyval,
                     void *extra_state, void *attribute_val_in,
                     void *attribute_val_out, int *flag) {
    NOT_IMPLEMENTED;
    return MPI_SUCCESS;
}

__device__ int MPI_NULL_DELETE_FN(MPI_Comm comm, int keyval,
                       void *attribute_val, void *extra_state) {
    NOT_IMPLEMENTED;
    return MPI_SUCCESS;
}

__device__ int MPI_Keyval_create(MPI_Copy_function *copy_fn,
                                 MPI_Delete_function *delete_fn, int *keyval, void *extra_state) {
    NOT_IMPLEMENTED;
    return MPI_SUCCESS;
}

__device__ int MPI_Dims_create(int nnodes, int ndims, int dims[]) {
    NOT_IMPLEMENTED;
    return MPI_SUCCESS;
}

__device__ int MPI_Irecv(void *buf, int count, MPI_Datatype datatype,
               int source, int tag, MPI_Comm comm, MPI_Request *request)
{
    int ctx = gpu_mpi::getCommContext(comm);
    
    int dataSize = gpu_mpi::TypeSize(datatype) * count;
    assert(dataSize > 0);
    
    CudaMPI::PendingOperation* op = CudaMPI::irecv(source, buf, dataSize, ctx, tag);
    
    if (request) {
        *request = new MPI_Request_impl(op);
    }
    
    return MPI_SUCCESS;
}
__device__ int MPI_Isend(const void *buf, int count, MPI_Datatype datatype, int dest,
                         int tag, MPI_Comm comm, MPI_Request *request) 
{
    int ctx = gpu_mpi::getCommContext(comm);
    
    int dataSize = gpu_mpi::TypeSize(datatype) * count;
    assert(dataSize > 0);
    
    CudaMPI::PendingOperation* op = CudaMPI::isend(dest, buf, dataSize, ctx, tag);
    
    *request = new MPI_Request_impl(op);
    return MPI_SUCCESS;
}

__device__ int MPI_Testall(int count, MPI_Request array_of_requests[],
            int *flag, MPI_Status array_of_statuses[]) {
    NOT_IMPLEMENTED;
    return MPI_SUCCESS;
}

__device__ int MPI_Waitall(int count, MPI_Request array_of_requests[],
            MPI_Status *array_of_statuses) {
    for (int i = 0; i < count; i++) {
        MPI_Wait(&array_of_requests[i], &array_of_statuses[i]);
    }
    return MPI_SUCCESS;
}

__device__ int MPI_Initialized(int *flag) {
    NOT_IMPLEMENTED;
    return MPI_SUCCESS;
}

__device__ int MPI_Waitsome(int incount, MPI_Request array_of_requests[],
            int *outcount, int array_of_indices[],
            MPI_Status array_of_statuses[]) {
    NOT_IMPLEMENTED;
    return MPI_SUCCESS;
}

__device__ int MPI_Wait(MPI_Request *request, MPI_Status *status) {
    if (request == MPI_REQUEST_NULL) {
        if (status) *status = MPI_Status();
    }
    
    switch ((*request)->type) {
        case MPI_Request_impl::Type::SR:
            CudaMPI::wait((*request)->pendingOperation);
            MPI_Request_free(request);
            if (status) *status = MPI_Status();
            break;
        case MPI_Request_impl::Type::IO:
            CudaMPI::waitIO((*request)->pendingIOOperation);
            MPI_Request_free(request);
            if (status) *status = MPI_Status();// ?
            break;
    }

    return MPI_SUCCESS;
}



__device__ int MPI_Request_free(MPI_Request *request) {
    switch ((*request)->type) {
        case MPI_Request_impl::Type::SR:
            assert((*request)->ref_count > 0);
            (*request)->ref_count--;
            if ((*request)->ref_count == 0) delete *request;
            *request = MPI_REQUEST_NULL;
            break;
        case MPI_Request_impl::Type::IO:
            assert((*request)->ref_count > 0);
            (*request)->ref_count--;
            if ((*request)->ref_count == 0) delete *request;
            *request = MPI_REQUEST_NULL;
            break;
    }
    return MPI_SUCCESS;
}

struct MPI_File;
/* ----- Non-blocking IO ------ */
#if USE_AIO && !USE_URING
__device__ int MPI_File_iread(MPI_File fh, void *buf, int count, MPI_Datatype datatype, MPI_Request *request){
    if (!(fh.amode & MPI_MODE_RDONLY) && !(fh.amode & MPI_MODE_RDWR)) return MPI_ERR_AMODE;
    if (fh.amode & MPI_MODE_SEQUENTIAL) return MPI_ERR_UNSUPPORTED_OPERATION;  // p514 l43
    int rank,size;
    MPI_Comm_rank(fh.comm, &rank);
    MPI_Comm_size(fh.comm, &size);
    if(rank == 0) {
        int msg_size = 48;  // sizeof(int) + sizeof(r_param) + sizeof(char) * (count + 1);: misaligned address
        int buffer_size = count*datatype.size();
        void *msg =  CudaMPI::sharedState().freeManagedMemory.allocate(msg_size);
        void *data = CudaMPI::sharedState().freeManagedMemory.allocate(buffer_size);
        char *p = (char*)msg;
        *((int*)p) = I_FILE_IREAD;          p += 8;
        *((FILE**)p) = fh.file;             p += 8;
        *((size_t*)p) = buffer_size;        p += 8;
        *((off_t*)p) = fh.seek_pos[rank];   p += 8;
        *((void**)p) = data;                p += 8; //aio temp buf, this is managed memory
        // *((void**)p) = buf;                 //dest buf, this is in process private space


        CudaMPI::sharedState().deviceToHostCommunicator.delegateToHost(msg, msg_size);  //schedule io task
        /* // debug only:
        // while (*((int*)msg) != I_READY){}
        // memcpy(buf, p, datatype.size() * count);
        */
        
        aiocb* newcb_p = new aiocb; //TODO: delete

        size_t ret = ((size_t*)msg)[1]; //ret of scheduling
        p = (char*)msg+8; 
        memcpy(newcb_p,(aiocb*)p,sizeof(aiocb));
        // *newcb_p = *((aiocb*)p); //get the cb
        //ori:: CudaMPI::PendingOperation* op = CudaMPI::iread(newcb_p);
        CudaMPI::PendingIOOperation* ioop = new CudaMPI::PendingIOOperation;
        ioop->aiocb_p = newcb_p;
        ioop->buf = buf;
        (*request) = new (MPI_Request_impl)(ioop, 1); //TODO: delete

        MPI_File_seek(fh, newcb_p->aio_nbytes, MPI_SEEK_CUR);// documentation P518 L30
        CudaMPI::sharedState().freeManagedMemory.free(msg);
        return ret;
    }
    return true;
}


#elif USE_URING
__device__ int MPI_File_iread(MPI_File fh, void *buf, int count, MPI_Datatype datatype, MPI_Request *request){}
#elif !USE_AIO
__device__ int MPI_File_iread(MPI_File fh, void *buf, int count, MPI_Datatype datatype, MPI_Request *request){}
#endif



