#include "hip/hip_runtime.h"
#include "io.cuh"
#include "../gpu_main/device_host_comm.cuh"
#include <cassert>

// #include "mpi.cuh"
#define N 100
namespace gpu_mpi {
}
    __device__ FILE* __open_file(const char* filename, int mode){
        if(filename == NULL){
            return nullptr;
        }
        int buffer_size = 128;
        char* data = (char*) allocate_host_mem(buffer_size);
        ((int*)data)[0] = I_FOPEN;
        ((int*)data)[1] = mode;

        int filename_size = 0;
        while (filename[filename_size] != '\0') filename_size++;
        memcpy((const char**)data + 2 , filename, filename_size+1);
        
        delegate_to_host((void*)data, buffer_size);
        // wait
        while(((int*)data)[0] != I_READY){};
        
        FILE* file = ((FILE**)data)[1];
        free_host_mem(data);
        return file;
    }

    __device__ void __close_file(FILE* file){
        if(file == NULL){
            return;
        }

        int buffer_size = 128;
        char* data = (char*) allocate_host_mem(buffer_size);
        // close the file associated with file handle
        ((int*)data)[0] = I_FCLOSE;
        ((FILE**)data)[1] = file;
        delegate_to_host((void*)data, buffer_size);
        // wait
        while(((int*)data)[0] != I_READY){};
        //fclose done
        free_host_mem(data);
    }

    __device__ long int __get_file_size(FILE* file){
        int buffer_size = 128;
        char* data = (char*) allocate_host_mem(buffer_size);
        ((int*)data)[0] = I_FSEEK;
        ((FILE**)data)[1] = file;
        delegate_to_host((void*)data, buffer_size);
        // wait
        while(((int*)data)[0] != I_READY){};
        long int file_length = ((long int*)data)[1];
        free_host_mem(data);
        return file_length;
    }


    __device__ int MPI_File_open(MPI_Comm comm, const char *filename, int amode, MPI_Info info, MPI_File *fh){
        __device__ __shared__ int err_code;
        __device__ __shared__ MPI_File shared_fh;

        // check amode
        if (((amode & MPI_MODE_RDONLY) ? 1 : 0) + ((amode & MPI_MODE_RDWR) ? 1 : 0) +
            ((amode & MPI_MODE_WRONLY) ? 1 : 0) != 1) {
            // see documentation p495 line 7
            err_code = MPI_ERR_AMODE;
        }
        if(((amode & MPI_MODE_CREATE) || (amode & MPI_MODE_EXCL)) && (amode & MPI_MODE_RDONLY)){
            // see documentation p495 line 8
            err_code = MPI_ERR_AMODE;
        }
        if((amode & MPI_MODE_RDWR) && (amode & MPI_MODE_SEQUENTIAL)){
            // see documentation p495 line 9
            err_code = MPI_ERR_AMODE;
        }

        // create MPI_FILE
        int rank;
        MPI_Comm_rank(comm, &rank);
        if(err_code == 0 && rank == 0){
            shared_fh.amode = amode;
            shared_fh.comm = comm;

            // initialize fh->file
            shared_fh.file = __open_file(filename, I_FOPEN_MODE_RD);

            // check file existence
            if(shared_fh.file == NULL){
                if(amode & MPI_MODE_RDONLY){
                    err_code = MPI_ERR_NO_SUCH_FILE;
                }
                if(!(amode & MPI_MODE_CREATE)){
                    err_code = MPI_ERR_NO_SUCH_FILE;
                }
            }
            if(amode & MPI_MODE_EXCL){
                // File must not exist
                err_code = MPI_ERR_FILE_EXISTS;
            }

            if(err_code != 0) {
                __close_file(shared_fh.file);
                __syncthreads();
                return err_code;
            }

            if(!(amode & MPI_MODE_RDONLY)){
                __close_file(shared_fh.file);
                int mode;
                if(amode & MPI_MODE_RDWR){
                    if(amode & MPI_MODE_APPEND) {
                        mode = I_FOPEN_MODE_RW_APPEND;
                    }else{
                        mode = I_FOPEN_MODE_RW;
                    }
                }else if(amode & MPI_MODE_WRONLY){
                    if(amode & MPI_MODE_APPEND) {
                        mode = I_FOPEN_MODE_WD_APPEND;
                    }else{
                        mode = I_FOPEN_MODE_WD;
                    }
                }
                shared_fh.file = __open_file(filename, mode);
            }
            
            // initialize fh->seek_pos
            // TODO: MPI_MODE_UNIQUE_OPEN -> Only one seek_pos???
            int size;
            MPI_Comm_rank(comm, &size);
            fh->seek_pos = (int*)malloc(size*sizeof(int));
            int init_pos = 0;
            if(amode & MPI_MODE_APPEND){
                // In append mode: set pointer to end of file 
                // see documentation p494 line 42
                init_pos = __get_file_size(shared_fh.file);
            }
            memset(shared_fh.seek_pos, init_pos, sizeof(int) * size);
        }
        
        __syncthreads();
        *fh = shared_fh;

        return err_code;
    }

    __device__ int MPI_File_seek(MPI_File fh, MPI_Offset offset, int whence){
        if(fh.amode & MPI_MODE_SEQUENTIAL){
            return MPI_ERR_UNSUPPORTED_OPERATION;
        }

        int rank;
        MPI_Comm_rank(fh.comm, &rank);
        if(whence == MPI_SEEK_SET){
            if(offset < 0){
                // see documentation p521 line 11
                return MPI_ERR_UNSUPPORTED_OPERATION;
            }
            fh.seek_pos[rank] = offset;
        }else if(whence == MPI_SEEK_CUR){
            int new_offset = fh.seek_pos[rank] + offset;
            if(new_offset < 0){
                // see documentation p521 line 11
                return MPI_ERR_UNSUPPORTED_OPERATION;
            }
            fh.seek_pos[rank] = new_offset;
        }else if(whence == MPI_SEEK_END){
            int sz = __get_file_size(fh.file);
            int new_offset = sz + offset;
            if(new_offset < 0){
                // see documentation p521 line 11
                return MPI_ERR_UNSUPPORTED_OPERATION;
            }
            fh.seek_pos[rank] = new_offset;
        }

        return 0;
    }

    __device__ int MPI_File_get_position(MPI_File fh, MPI_Offset *offset){
        int rank;
        MPI_Comm_rank(fh.comm, &rank);
        *offset = fh.seek_pos[rank];
        return 0;
    }

    __device__ int MPI_File_read(MPI_File fh, void *buf, int count, MPI_Datatype datatype, MPI_Status *status){
        if (!(fh.amode & MPI_MODE_RDONLY) && !(fh.amode & MPI_MODE_RDWR)) return MPI_ERR_AMODE;
        if (fh.amode & MPI_MODE_SEQUENTIAL) return MPI_ERR_UNSUPPORTED_OPERATION;  // p514 l43
        // TODO: Only one thread with RDWR can gain access; unlimited threads with RDONLY can gain access (?)
        // TODO: write into MPI_Status

        int buffer_size = sizeof(int) + sizeof(FILE*) + sizeof(MPI_Datatype) + sizeof(void*) + sizeof(int) + 2048;  // (TODO: dynamic size) sizeof(datatype) * count;
        void* data = (void*)allocate_host_mem(buffer_size);
        ((int*)data)[0] = I_FREAD;
        ((FILE**)data)[1] = fh.file;
        ((MPI_Datatype*)data)[2] = datatype;
        ((void**)data)[3] = (void**)data + 5;  // buf;
        ((int*)data)[4] = count;
        
        delegate_to_host((void*)data, buffer_size);
        while (((int*)data)[0] != I_READY)
        {
            // blocking wait (p506 l44)
        }

        memcpy(buf, (void**)data + 5, sizeof(datatype) * count);
        free_host_mem(data);
        return ((size_t*)data)[1];
    }
    
    __device__ int __howManyBits(int x) {
        assert(sizeof(int)==4);
        return __double2int_ru(__log2f(x));
        // int a = x >> 31;
        // int newx = (x ^ a); //2
        // int temp1 = !!(newx >> 16);//5
        // int ntemp1 = ~temp1 + 1; //7
        // int n = 16 & ntemp1;     //8
        // int temp2 = !!(newx >> (8 + n)); //12
        // int ntemp2 = ~temp2 + 1; //14
        // int temp3 = 0;
        // int ntemp3 = 0;
        // int temp4 = 0;
        // int ntemp4 = 0;
        // int temp5 = 0;

        // n = n + (ntemp2 & 8);
        // temp3 = !!(newx >> (4 + n));
        // ntemp3 = ~temp3 + 1;
        // n = n + (ntemp3 & 4);
        // temp4 = !!(newx >> (2 + n));
        // ntemp4 = ~temp4 + 1;
        // n = n + (ntemp4 & 2);
        // temp5 = !!(newx >> (1 + n));
        // n = n + temp5 + 1 + !!newx; 
        // return n;
    }

    //for debug
    __device__ __host__ void __show_memory(char * mem, size_t size){
        char *tmem = (char *)mem;
        for(int i=0;i+7<size;i+=8){
            printf("%02X  %02X  %02X  %02X  %02X  %02X  %02X  %02X\n",tmem[i],tmem[i+1],tmem[i+2],tmem[i+3],tmem[i+4],tmem[i+5],tmem[i+6],tmem[i+7]);
        }
    }

    //not thread safe
    __device__ int MPI_File_write(MPI_File fh, const void *buf, int count, MPI_Datatype datatype, MPI_Status *status){
        //TODO: dynamically assign buffer size
        int buffer_size = 2048;
        // int MPI_Type_size(MPI_Datatype datatype, int *size)
        //TODO: MPI_Type_size not implemented
        assert(datatype==MPI_CHAR);
        assert(buffer_size > sizeof(int*)*2+sizeof(FILE**)+sizeof(char)*count);
        //init
        char* data = (char*) allocate_host_mem(buffer_size);
        //assemble
        *((int*)data) = I_FWRITE;
        *((int*)(data+4)) = count;
        // printf("OS file descripter address in MPI_File_write:%p\n", fh.file);
        *((FILE **)(data+8)) = fh.file;
        // __show_memory(data, 64);
        

        *((MPI_Datatype*)(data+16)) = datatype;
        memcpy( ((const char**)data+24) , buf, sizeof(char)*count);

        //execute on CPU
        delegate_to_host((void*)data, buffer_size);
        // wait
        while(((int*)data)[0] != I_READY){};
        int return_value = (int) *((size_t*)(data+8));
        
        int rank;
        MPI_Comm_rank(fh.comm, &rank);
        //assuming individual file pointer, but how does shared pointer differ from this?
        // fh.seek_pos[rank]+=return_value;
        free_host_mem(data);
        //TODO: step 4 error catching
        //#memory cosistency: assuming that write is not reordered with write
        return return_value;
    }


    __device__ int MPI_File_close(MPI_File *fh){
        // synchronize file state
        __syncthreads();

        int rank;
        MPI_Comm_rank(fh->comm, &rank);

        // only free the file handle object once
        if(rank == 0){
            // close the file associated with file handle
            // fclose(fh->file);
            __close_file(fh->file);
            
            // release the fh object
            free(fh->seek_pos);
        }
        __syncthreads();
        //MPI_Barrier(MPI_COMM_WORLD);
        return 0;
    }

