#include "communicator.cuh"

#include "mpi_common.cuh"
#include "group.cuh"

#include <hip/hip_cooperative_groups.h>
using namespace cooperative_groups;

#include "cuda_mpi.cuh"

#include "mpi.h.cuh"

#include "assert.h.cuh"

#include "stdlib.cuh"

struct MPI_Comm_impl {
    int context;
    MPI_Group group;
};


__device__ MPI_Comm MPI_COMM_WORLD = (MPI_Comm)nullptr;
__device__ MPI_Comm MPI_COMM_NULL = (MPI_Comm)nullptr;

namespace gpu_mpi {

__device__ int createNewContextId(MPI_Comm comm) {
    int freeCommCtx = CudaMPI::threadPrivateState().unusedCommunicationContext;
    MPI_Allreduce(&freeCommCtx, &freeCommCtx, 1, MPI_INT, MPI_MAX, comm);
    CudaMPI::threadPrivateState().unusedCommunicationContext = freeCommCtx + 1;
    if (freeCommCtx > (1 << 15)) {
        printf("ERROR: the limit of communicator contexts is exceeded!\n");
        __gpu_abort();
    }
    return freeCommCtx;
}
    
__device__ int getCommContext(MPI_Comm comm) {
    return comm->context;
};

__device__ void initializeGlobalCommunicators() {
    if (this_grid().thread_rank() == 0) {
        MPI_COMM_NULL = new MPI_Comm_impl;
        MPI_COMM_NULL->context = 0;
        MPI_COMM_NULL->group = MPI_GROUP_EMPTY;
        
        MPI_COMM_WORLD = new MPI_Comm_impl;
        MPI_COMM_WORLD->context = 1;
        MPI_COMM_WORLD->group = MPI_GROUP_WORLD;
        
        CudaMPI::threadPrivateState().unusedCommunicationContext = 2;
    }
    this_grid().sync();
}

} // namespace

__device__ int MPI_Comm_free(MPI_Comm *comm) {
    delete *comm;
    return MPI_SUCCESS;
}

__device__ int MPI_Comm_group(MPI_Comm comm, MPI_Group *group) {
    *group = comm->group;
    return MPI_SUCCESS;
}

__device__ int MPI_Comm_create(
    MPI_Comm comm, MPI_Group group, MPI_Comm *newcomm) 
{
    int ctxId = gpu_mpi::createNewContextId(comm);
    
    int rank = MPI_UNDEFINED;
    MPI_Group_rank(group, &rank);
    if (rank == MPI_UNDEFINED) {
        *newcomm = MPI_COMM_NULL;
        return MPI_SUCCESS;
    } else {
        MPI_Comm commImpl = new MPI_Comm_impl;
        commImpl->context = ctxId;
        commImpl->group = group;
        *newcomm = commImpl;
        return MPI_SUCCESS;
    }
}

__device__ int MPI_Attr_put(MPI_Comm comm, int keyval, void *attribute_val) {
    return MPI_SUCCESS;
}

__device__ int MPI_Attr_get(MPI_Comm comm, int keyval,void *attribute_val, int *flag) {
    return MPI_SUCCESS;
}

__device__ int MPI_Cart_create(MPI_Comm comm_old, int ndims, const int dims[], const int periods[], int reorder, MPI_Comm *comm_cart) {
    MPI_Comm commImpl = new MPI_Comm_impl;
    
    int ctxId = gpu_mpi::createNewContextId(comm_old);
    
    commImpl->context = ctxId;
    commImpl->group = comm_old->group;
    
    *comm_cart = commImpl;
    
    return MPI_SUCCESS;
}

__device__ int MPI_Cart_sub(MPI_Comm comm, const int remain_dims[], MPI_Comm *comm_new) {
    NOT_IMPLEMENTED
    return MPI_SUCCESS;
}

__device__ int MPI_Comm_split(MPI_Comm comm, int color, int key, MPI_Comm *newcomm) {
    NOT_IMPLEMENTED
    return MPI_SUCCESS;
}

__device__ int MPI_Comm_size(MPI_Comm comm, int *size) {
    return MPI_Group_size(comm->group, size);
}

__device__ int MPI_Comm_rank(MPI_Comm comm, int *rank) {
    return MPI_Group_rank(comm->group, rank);
}


