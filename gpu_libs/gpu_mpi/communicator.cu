#include "communicator.cuh"

#include "mpi_common.cuh"
#include "group.cuh"

#include <hip/hip_cooperative_groups.h>
using namespace cooperative_groups;

#include "cuda_mpi.cuh"

#include "mpi.h.cuh"

#include "assert.h.cuh"

#include "stdlib.cuh"

#include <memory>

struct MPI_Comm_impl {
    __device__ MPI_Comm_impl(int context, MPI_Group group) 
        : context(context), group(group), ref_count(1)
    {
        gpu_mpi::incGroupRefCount(group);
    }
    
    __device__ ~MPI_Comm_impl() {
        MPI_Group_free(&group);
    }
    
    int context;
    MPI_Group group;
    int ref_count;
};

__device__ MPI_Comm MPI_COMM_WORLD = (MPI_Comm)nullptr;
__device__ MPI_Comm MPI_COMM_NULL = (MPI_Comm)nullptr;

namespace gpu_mpi {

__device__ int createNewContextId(MPI_Comm comm) {
    int freeCommCtx = CudaMPI::threadPrivateState().unusedCommunicationContext;
    MPI_Allreduce(&freeCommCtx, &freeCommCtx, 1, MPI_INT, MPI_MAX, comm);
    CudaMPI::threadPrivateState().unusedCommunicationContext = freeCommCtx + 1;
    if (freeCommCtx > (1 << 15)) {
        printf("ERROR: the limit of communicator contexts is exceeded!\n");
        __gpu_abort();
    }
    return freeCommCtx;
}
    
__device__ int getCommContext(MPI_Comm comm) {
    return comm->context;
};

__device__ void initializeGlobalCommunicators() {
    if (this_grid().thread_rank() == 0) {
        MPI_COMM_NULL = new MPI_Comm_impl(0, MPI_GROUP_EMPTY);
        
        MPI_COMM_WORLD = new MPI_Comm_impl(1, MPI_GROUP_WORLD);
        
    }
    this_grid().sync();

    CudaMPI::threadPrivateState().unusedCommunicationContext = 2;
}

__device__ void destroyGlobalCommunicators() {
    this_grid().sync();
    if (this_grid().thread_rank() == 0) {
        delete MPI_COMM_NULL;
        delete MPI_COMM_WORLD;
    }
}

__device__ void incCommRefCount(MPI_Comm comm) {
    assert(comm->ref_count > 0);
    
    if (comm == MPI_COMM_WORLD || comm == MPI_COMM_NULL) return;
    
    comm->ref_count += 1;
}

} // namespace

__device__ int MPI_Comm_free(MPI_Comm *comm) {
    assert((*comm)->ref_count > 0);
    
    if (*comm == MPI_COMM_WORLD || *comm == MPI_COMM_NULL) {
        *comm = MPI_COMM_NULL;
        return MPI_SUCCESS;
    }
    
    (*comm)->ref_count--;
    if ((*comm)->ref_count == 0) delete *comm;
    *comm = MPI_COMM_NULL;
    return MPI_SUCCESS;
}

__device__ int MPI_Comm_group(MPI_Comm comm, MPI_Group *group) {
    *group = comm->group;
    gpu_mpi::incGroupRefCount(*group);
    return MPI_SUCCESS;
}

__device__ int MPI_Comm_create(
    MPI_Comm comm, MPI_Group group, MPI_Comm *newcomm) 
{
    int ctxId = gpu_mpi::createNewContextId(comm);
    
    int rank = MPI_UNDEFINED;
    MPI_Group_rank(group, &rank);
    if (rank == MPI_UNDEFINED) {
        *newcomm = MPI_COMM_NULL;
        return MPI_SUCCESS;
    } else {
        *newcomm = new MPI_Comm_impl(ctxId, group);
        return MPI_SUCCESS;
    }
}

__device__ int MPI_Comm_dup(MPI_Comm comm, MPI_Comm *newcomm) {
    if (comm == MPI_COMM_NULL) {
       return MPI_FAILURE; 
    } else {
        int ctxId = gpu_mpi::createNewContextId(comm);
        MPI_Group group = comm->group;
        gpu_mpi::incGroupRefCount(comm->group);
        *newcomm = new MPI_Comm_impl(ctxId, group);
        return MPI_SUCCESS;
    }
}

__device__ int MPI_Attr_put(MPI_Comm comm, int keyval, void *attribute_val) {
    return MPI_SUCCESS;
}

__device__ int MPI_Attr_get(MPI_Comm comm, int keyval,void *attribute_val, int *flag) {
    return MPI_SUCCESS;
}

__device__ int MPI_Cart_create(MPI_Comm comm_old, int ndims, const int dims[], const int periods[], int reorder, MPI_Comm *comm_cart) {
    NOT_IMPLEMENTED
    return MPI_SUCCESS;
}

__device__ int MPI_Cart_sub(MPI_Comm comm, const int remain_dims[], MPI_Comm *comm_new) {
    NOT_IMPLEMENTED
    return MPI_SUCCESS;
}

__device__ int MPI_Comm_split(MPI_Comm comm, int color, int key, MPI_Comm *newcomm) {
    
    return MPI_SUCCESS;
}

__device__ int MPI_Comm_size(MPI_Comm comm, int *size) {
    return MPI_Group_size(comm->group, size);
}

__device__ int MPI_Comm_rank(MPI_Comm comm, int *rank) {
    return MPI_Group_rank(comm->group, rank);
}


