#include "hip/hip_runtime.h"
#include "test_runner.cuh"

#include "mpi.h.cuh"

namespace cg = cooperative_groups;

struct SingleIntKernel {
    static __device__ void run(bool& ok)
    {
        MPI_Init(nullptr, nullptr);
        int rank = -1;
        MPI_CHECK_DEVICE(MPI_Comm_rank(MPI_COMM_WORLD, &rank));

        if (rank == 0) {
            int x = 3456;

            CudaMPI::PendingOperation* op = CudaMPI::isend(1, &x, sizeof(int), 0, 15);

            CudaMPI::wait(op);

            ok = true;
        } else if (rank == 1) {
            int x = 0;

            CudaMPI::PendingOperation* op = CudaMPI::irecv(0, &x, sizeof(int), 0, 15);

            CudaMPI::wait(op);

            ok = x == 3456;
        }
        MPI_Finalize();
    }
};

TEST_CASE("Transfer single integer", "[single_int]") {
    TestRunner testRunner(2);
    testRunner.run<SingleIntKernel>();
}


__global__ void transfer_array_kernel(
    CudaMPI::SharedState* sharedState,
    CudaMPI::ThreadPrivateState::Context threadPrivateStateContext,
    bool* ok)
{
    CudaMPI::setSharedState(sharedState);
    CudaMPI::ThreadPrivateState::Holder threadPrivateStateHolder(threadPrivateStateContext);

    if (cg::this_grid().thread_rank() == 0) {
        int x[16] = {0, 1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 11, 12, 13, 14, 15};

        CudaMPI::PendingOperation* op = CudaMPI::isend(1, &x, sizeof(x), 0, 15);

        CudaMPI::wait(op);
    } else if (cg::this_grid().thread_rank() == 1) {
        int x[16] = {};

        CudaMPI::PendingOperation* op = CudaMPI::irecv(0, &x, sizeof(x), 0, 15);

        CudaMPI::wait(op);

        *ok = true;
        for (int i = 0; i < 16; i++) {
            if (x[i] != i) *ok = false;
        }
    }
}

TEST_CASE("Transfer array", "[array]") {
    CudaMPI::SharedState::Context sharedStateContext = {2, 10, 10, 10, 10};
    CudaMPI::SharedState::Holder sharedStateHolder(sharedStateContext);
    CudaMPI::ThreadPrivateState::Context threadPrivateStateContext;
    threadPrivateStateContext.peakClockKHz = 100;

    bool* ok;
    CUDA_CHECK(hipMallocManaged(&ok, sizeof(bool)));
    *ok = 0;

    CudaMPI::SharedState* sharedStatePtr = sharedStateHolder.get();
    
    void* params[] = {
        (void*)&sharedStatePtr, 
        (void*)&threadPrivateStateContext,
        (void*)&ok
    };
    
    CUDA_CHECK(hipLaunchCooperativeKernel((void*)transfer_array_kernel, 3, 1, params));
    CUDA_CHECK(hipPeekAtLastError());
    CUDA_CHECK(hipDeviceSynchronize());

    REQUIRE(*ok == true);
}

__global__ void send_recv_kernel(
    CudaMPI::SharedState* sharedState,
    CudaMPI::ThreadPrivateState::Context threadPrivateStateContext,
    bool* ok)
{
    CudaMPI::setSharedState(sharedState);
    CudaMPI::ThreadPrivateState::Holder threadPrivateStateHolder(threadPrivateStateContext);

    if (cg::this_grid().thread_rank() == 0) {
        int x[16] = {0, 1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 11, 12, 13, 14, 15};
        int y[16] = {};

        CudaMPI::PendingOperation* op[2];
        op[0] = CudaMPI::irecv(1, &y, sizeof(y), 0, 15);
        op[1] = CudaMPI::isend(1, &x, sizeof(x), 0, 15);

        CudaMPI::wait(op[0]);

        ok[0] = true;
        for (int i = 0; i < 16; i++) {
            if (y[i] != -i) {
                printf("thread %d, y[i] = %d\n", cg::this_grid().thread_rank(), y[i]);
                ok[0] = false;
            }
        }

        CudaMPI::wait(op[1]);
    } else if (cg::this_grid().thread_rank() == 1) {
        int x[16] = {0, -1, -2, -3, -4, -5, -6, -7, -8, -9, -10, -11, -12, -13, -14, -15};
        int y[16] = {};

        CudaMPI::PendingOperation* op[2];
        op[0] = CudaMPI::irecv(0, &y, sizeof(y), 0, 15);
        op[1] = CudaMPI::isend(0, &x, sizeof(x), 0, 15);

        CudaMPI::wait(op[0]);

        ok[1] = true;
        for (int i = 0; i < 16; i++) {
            if (y[i] != i) {
                printf("thread %d, y[i] = %d\n", cg::this_grid().thread_rank(), y[i]);
                ok[1] = false;
            }
        }

        CudaMPI::wait(op[1]);
    }
}

TEST_CASE("Send receive", "[send_recv]") {
    CudaMPI::SharedState::Context sharedStateContext = {2, 10, 10, 10, 10};
    CudaMPI::SharedState::Holder sharedStateHolder(sharedStateContext);
    CudaMPI::ThreadPrivateState::Context threadPrivateStateContext;
    threadPrivateStateContext.peakClockKHz = 100;

    bool* ok;
    CUDA_CHECK(hipMallocManaged(&ok, 2 * sizeof(bool)));
    ok[0] = false;
    ok[1] = false;

    CudaMPI::SharedState* sharedStatePtr = sharedStateHolder.get();
    
    void* params[] = {
        (void*)&sharedStatePtr, 
        (void*)&threadPrivateStateContext,
        (void*)&ok
    };
    
    CUDA_CHECK(hipLaunchCooperativeKernel((void*)send_recv_kernel, 3, 1, params));
    CUDA_CHECK(hipPeekAtLastError());
    CUDA_CHECK(hipDeviceSynchronize());

    REQUIRE(ok[0] == true);
    REQUIRE(ok[1] == true);
}

__global__ void repeat_sendrecv_kernel(
    CudaMPI::SharedState* sharedState,
    CudaMPI::ThreadPrivateState::Context threadPrivateStateContext,
    bool* ok)
{
    CudaMPI::setSharedState(sharedState);
    CudaMPI::ThreadPrivateState::Holder threadPrivateStateHolder(threadPrivateStateContext);

    const int numRanks = 2;
    
    int thisRank = cg::this_grid().thread_rank();
    int otherRank = (thisRank + 1) % numRanks;
    
    const int numRepeats = 5;
    const int dataSize = 1 << (numRepeats - 1);
    
    int localData[dataSize] = {};
    for (int i = 0; i < dataSize; i++) {
        localData[i] = (thisRank + 1) * i;
    }
    
    int remoteData[dataSize] = {};
    
    CudaMPI::PendingOperation* send_op[numRepeats];
    CudaMPI::PendingOperation* recv_op[numRepeats];
    
    for (int i = 0; i < numRepeats; i++) {
        int tag = i + 10;
        send_op[i] = CudaMPI::isend(otherRank, localData, sizeof(int) * (1 << i), 0, tag);
        recv_op[i] = CudaMPI::irecv(otherRank, remoteData, sizeof(int) * (1 << i), 0, tag);
    }
    
    for (int i = 0; i < numRepeats; i++) {
        CudaMPI::wait(send_op[i]);
        CudaMPI::wait(recv_op[i]);
    }
    
    ok[thisRank] = true;
    for (int i = 0; i < dataSize; i++) {
        if (remoteData[i] != (otherRank + 1) * i) ok[thisRank] = false;
    }
}

TEST_CASE("Repeat send recv", "[repeat_sendrecv]") {
    CudaMPI::SharedState::Context sharedStateContext = {2, 10, 10, 10, 10};
    CudaMPI::SharedState::Holder sharedStateHolder(sharedStateContext);
    CudaMPI::ThreadPrivateState::Context threadPrivateStateContext;
    threadPrivateStateContext.peakClockKHz = 100;

    bool* ok;
    CUDA_CHECK(hipMallocManaged(&ok, 2 * sizeof(bool)));
    ok[0] = false;
    ok[1] = false;

    CudaMPI::SharedState* sharedStatePtr = sharedStateHolder.get();
    
    void* params[] = {
        (void*)&sharedStatePtr, 
        (void*)&threadPrivateStateContext,
        (void*)&ok
    };
    
    CUDA_CHECK(hipLaunchCooperativeKernel((void*)repeat_sendrecv_kernel, 2, 1, params));
    CUDA_CHECK(hipPeekAtLastError());
    CUDA_CHECK(hipDeviceSynchronize());

    REQUIRE(ok[0] == true);
    REQUIRE(ok[1] == true);
}

__global__ void network_flood_kernel(
    CudaMPI::SharedState* sharedState,
    CudaMPI::ThreadPrivateState::Context threadPrivateStateContext,
    bool* ok)
{
    CudaMPI::setSharedState(sharedState);
    CudaMPI::ThreadPrivateState::Holder threadPrivateStateHolder(threadPrivateStateContext);

    const int numRanks = 3;
    
    int thisRank = cg::this_grid().thread_rank();
    int nextRank = (thisRank + 1) % numRanks;
    int prevRank = (numRanks + thisRank - 1) % numRanks;
    
    const int numRepeats = 5;
    const int dataSize = 1 << (numRepeats - 1);
    
    int localData[dataSize] = {};
    for (int i = 0; i < dataSize; i++) {
        localData[i] = (thisRank + 1) * i;
    }
    
    int prevData[dataSize] = {};
    int nextData[dataSize] = {};
    
    
    CudaMPI::PendingOperation* send_next_op[numRepeats];
    CudaMPI::PendingOperation* recv_next_op[numRepeats];
    
    CudaMPI::PendingOperation* send_prev_op[numRepeats];
    CudaMPI::PendingOperation* recv_prev_op[numRepeats];
    
    for (int i = 0; i < numRepeats; i++) {
        int tag = i + 10;
        send_next_op[i] = CudaMPI::isend(nextRank, localData, sizeof(int) * (1 << i), 0, tag);
        recv_next_op[i] = CudaMPI::irecv(nextRank, nextData, sizeof(int) * (1 << i), 0, tag);
        send_prev_op[i] = CudaMPI::isend(prevRank, localData, sizeof(int) * (1 << i), 0, tag);
        recv_prev_op[i] = CudaMPI::irecv(prevRank, prevData, sizeof(int) * (1 << i), 0, tag);
    }
    
    for (int i = 0; i < numRepeats; i++) {
        CudaMPI::wait(recv_prev_op[i]);
        CudaMPI::wait(send_next_op[i]);
        CudaMPI::wait(send_prev_op[i]);
        CudaMPI::wait(recv_next_op[i]);
    }
    
    ok[thisRank] = true;
    for (int i = 0; i < dataSize; i++) {
        if (prevData[i] != (prevRank + 1) * i) ok[thisRank] = false;
        if (nextData[i] != (nextRank + 1) * i) ok[thisRank] = false;
    }
}

TEST_CASE("Network flood", "[network_flood]") {
    CudaMPI::SharedState::Context sharedStateContext = {3, 10, 10, 10, 10};
    CudaMPI::SharedState::Holder sharedStateHolder(sharedStateContext);
    CudaMPI::ThreadPrivateState::Context threadPrivateStateContext;
    threadPrivateStateContext.peakClockKHz = 100;

    bool* ok;
    CUDA_CHECK(hipMallocManaged(&ok, 3 * sizeof(bool)));
    ok[0] = false;
    ok[1] = false;
    ok[2] = false;

    CudaMPI::SharedState* sharedStatePtr = sharedStateHolder.get();
    
    void* params[] = {
        (void*)&sharedStatePtr, 
        (void*)&threadPrivateStateContext,
        (void*)&ok
    };
    
    CUDA_CHECK(hipLaunchCooperativeKernel((void*)network_flood_kernel, 3, 1, params));
    CUDA_CHECK(hipPeekAtLastError());
    CUDA_CHECK(hipDeviceSynchronize());

    REQUIRE(ok[0] == true);
    REQUIRE(ok[1] == true);
    REQUIRE(ok[2] == true);
}

__global__ void all_to_all_kernel(
    CudaMPI::SharedState* sharedState,
    CudaMPI::ThreadPrivateState::Context threadPrivateStateContext,
    int numRanks,
    bool* ok)
{
    CudaMPI::setSharedState(sharedState);
    CudaMPI::ThreadPrivateState::Holder threadPrivateStateHolder(threadPrivateStateContext);
    
    int thisRank = cg::this_grid().thread_rank();
    
    const int numRepeats = 20;
    const int dataSize = 16;
    
    int* localData = (int*) malloc(sizeof(int) * dataSize * numRepeats * numRanks);
    assert(localData);
    int* remoteData = (int*) malloc(sizeof(int) * dataSize * numRepeats * numRanks);
    assert(remoteData);
    
    for (int rank = 0; rank < numRanks; rank++) {
        for (int repeat = 0; repeat < numRepeats; repeat++) {
            for (int i = 0; i < dataSize; i++) {
                int idx = i + repeat * dataSize + rank * dataSize * numRepeats;
                localData[idx] = i * repeat * rank;
                remoteData[idx] = 0;
            }
        }
    }
    
    CudaMPI::PendingOperation** send_po = (CudaMPI::PendingOperation**) malloc(numRepeats * numRanks * sizeof(CudaMPI::PendingOperation*));
    CudaMPI::PendingOperation** recv_po = (CudaMPI::PendingOperation**) malloc(numRepeats * numRanks * sizeof(CudaMPI::PendingOperation*));
    
    int tag = 15;
    int comm = 17;
    
    for (int repeat = 0; repeat < numRepeats; repeat++) {
        for (int rank = 0; rank < numRanks; rank++) {
            if (rank != thisRank) {
                int idx = repeat * dataSize + rank * dataSize * numRepeats;
                send_po[rank + repeat * numRanks] = CudaMPI::isend(
                    rank, localData + idx, sizeof(int) * dataSize, comm, tag);
                recv_po[rank + repeat * numRanks] = CudaMPI::irecv(
                    rank, remoteData + idx, sizeof(int) * dataSize, comm, tag);
            }
        }
    }
    
    for (int repeat = 0; repeat < numRepeats; repeat++) {
        for (int rank = 0; rank < numRanks; rank++) {
            if (rank != thisRank) {
                CudaMPI::wait(send_po[rank + repeat * numRanks]);
                CudaMPI::wait(recv_po[rank + repeat * numRanks]);
            }
        }
    }
    
    ok[thisRank] = true;
    for (int repeat = 0; repeat < numRepeats; repeat++) {
        for (int rank = 0; rank < numRanks; rank++) {
            if (rank != thisRank) {
                for (int i = 0; i < dataSize; i++) {
                    int idx = i + repeat * dataSize + rank * dataSize * numRepeats;
                    if (i * repeat * thisRank != remoteData[idx]) {
                        ok[thisRank] = false;
                        printf("thisRank = %d, i = %d, repeat = %d, rank = %d, remoteData[%d] = %d\n", thisRank, i, repeat, rank, idx, remoteData[idx]);
                    }
                }
            }
        }
    }
    
    free(send_po);
    free(recv_po);
    free(localData);
}

TEST_CASE("All to all", "[all_to_all]") {
    const int numRanks = 10;
    
    CudaMPI::SharedState::Context sharedStateContext = {numRanks, 10, 10, 10, 10};
    CudaMPI::SharedState::Holder sharedStateHolder(sharedStateContext);
    CudaMPI::ThreadPrivateState::Context threadPrivateStateContext;
    threadPrivateStateContext.pendingBufferSize = 400;
    threadPrivateStateContext.peakClockKHz = 100;

    bool* ok;
    CUDA_CHECK(hipMallocManaged(&ok, numRanks * sizeof(bool)));
    for (int i = 0; i < numRanks; i++) {
        ok[i] = false;
    }

    CudaMPI::SharedState* sharedStatePtr = sharedStateHolder.get();
    
    void* params[] = {
        (void*)&sharedStatePtr,
        (void*)&threadPrivateStateContext,
        (void*)&numRanks,
        (void*)&ok
    };
    
    CUDA_CHECK(hipLaunchCooperativeKernel((void*)all_to_all_kernel, numRanks, 1, params));
    CUDA_CHECK(hipPeekAtLastError());
    CUDA_CHECK(hipDeviceSynchronize());

    for (int i = 0; i < numRanks; i++) {
        REQUIRE(ok[i] == true);
    }
}

